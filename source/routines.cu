#include "hip/hip_runtime.h"

#include "routines.hpp"
#include "ppt/execution/ExecutionSpacesInc.hpp"

using namespace ppt;

#define BLOCKDIM_X 64
#define BLOCKDIM_Z 1

__global__ void inject_source_kernel( 
    float_type *const p, 
    float_type amplitude,
    int iz, 
    int ix, 
    int nz, 
    int nx )
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if ( idx > 0 ) return;
    if ( iz >= nz || ix >= nx ) return;

    p[ iz * nx + ix ] = amplitude;
}

template<>
int inject_source( Fields<MemSpaceCuda> &fields, float_type amplitude, int iz, int ix, ExecutionSpaceCuda ){
    
    int const nz = fields.V->get_nz();
    int const nx = fields.V->get_nx();

    float_type *const p = fields.V->get_ptr();
    
    inject_source_kernel<<<1,1>>>( 
        p, amplitude, iz, ix, nz, nx 
    );

    return 0;
}

__global__ void velocity_update_kernel(
    float_type * const Vnew, 
    float_type const* const Vold, 
    float_type const* const Sold, 
    float_type const* const Told, 
    float_type const* const L, 
    int nz, 
    int nx,
    float_type dt_div_dz, 
    float_type dt_div_dx )
{
    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    int iz = blockDim.y * blockIdx.y + threadIdx.y;

    if ( ix >= nx - 1 ) return;
    if ( iz >= nz - 1 ) return;

    int idx = iz * nx + ix;
    Vnew[idx] = Vold[idx] + dt_div_dz * L[idx] * (Sold[(iz+1)*nx + ix] - Sold[idx]) + dt_div_dx * L[idx] * (Told[idx+1] - Told[idx]);
}

template<>
int velocity_update( Fields<MemSpaceCuda> &fields_new, Fields<MemSpaceCuda> const& fields_old, Models<MemSpaceCuda> const& model,
    float_type dt, float_type dz, float_type dx, ExecutionSpaceCuda ){
    int const nz = fields_new.V->get_nz();
    int const nx = fields_new.V->get_nx();

    float_type const* const Vold = fields_old.V->get_ptr();
    float_type const* const Sold = fields_old.S->get_ptr();
    float_type const* const Told = fields_old.T->get_ptr();
    float_type const* const L    = model.L->get_ptr();

    float_type const dt_div_dz = dt/dz;
    float_type const dt_div_dx = dt/dx;
    
    float_type * const Vnew = fields_new.V->get_ptr();

    dim3 nThreads(BLOCKDIM_X, BLOCKDIM_Z, 1);
    size_t nBlock_x = nx % BLOCKDIM_X == 0 ? size_t(nx / BLOCKDIM_X) : size_t(1 + nx / BLOCKDIM_X);
    size_t nBlock_z = nz % BLOCKDIM_Z == 0 ? size_t(nz / BLOCKDIM_Z) : size_t(1 + nz / BLOCKDIM_Z);
    dim3 nBlocks(nBlock_x, nBlock_z, 1);

    velocity_update_kernel<<<nBlocks,nThreads>>>(
        Vnew, Vold, Sold, Told, L, nz, nx, dt_div_dz, dt_div_dx 
    );

    return 0;
}


__global__ void stresses_update_kernel(
    float_type * const Snew, 
    float_type * const Tnew, 
    float_type const* const Vnew, 
    float_type const* const Sold, 
    float_type const* const Told, 
    float_type const* const M, 
    int nz, 
    int nx, 
    float_type dt_div_dz, 
    float_type dt_div_dx )
{
    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    int iz = blockDim.y * blockIdx.y + threadIdx.y;

    if ( ix == 0 || ix >= nx ) return;
    if ( iz == 0 || iz >= nz ) return;
    
    int idx = iz * nx + ix;
    Snew[idx] = Sold[idx] + dt_div_dz * M[idx] * (Vnew[idx] - Vnew[(iz-1)*nx+ix]);
    Tnew[idx] = Told[idx] + dt_div_dx * M[idx] * (Vnew[idx] - Vnew[idx-1]);
}

template<>
int stresses_update( Fields<MemSpaceCuda> &fields_new, Fields<MemSpaceCuda> const& fields_old, Models<MemSpaceCuda> const& model,
    float_type dt, float_type dz, float_type dx, ExecutionSpaceCuda ){
    int const nz = fields_new.V->get_nz();
    int const nx = fields_new.V->get_nx();

    float_type const* const Vnew = fields_new.V->get_ptr();
    float_type const* const Sold = fields_old.S->get_ptr();
    float_type const* const Told = fields_old.T->get_ptr();

    float_type const* const M    = model.M->get_ptr();

    float_type const dt_div_dz = dt/dz;
    float_type const dt_div_dx = dt/dx;

    float_type * const Snew = fields_new.S->get_ptr();
    float_type * const Tnew = fields_new.T->get_ptr();

    dim3 nThreads(BLOCKDIM_X, BLOCKDIM_Z, 1);
    size_t nBlock_x = nx % BLOCKDIM_X == 0 ? size_t(nx / BLOCKDIM_X) : size_t(1 + nx / BLOCKDIM_X);
    size_t nBlock_z = nz % BLOCKDIM_Z == 0 ? size_t(nz / BLOCKDIM_Z) : size_t(1 + nz / BLOCKDIM_Z);
    dim3 nBlocks(nBlock_x, nBlock_z, 1);

    stresses_update_kernel<<<nBlocks,nThreads>>>(
        Snew, Tnew, Vnew, Sold, Told, M, nz, nx, dt_div_dz, dt_div_dx
    );

    return 0;
}
